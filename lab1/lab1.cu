
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <malloc.h>
const int N = 1 << 20;

__global__ void gInitVectors(double* vector1, double* vector2) {
	for (int i = 0; i < N; i++) {
		vector1[i] = (double)i; //rand();
		vector2[i] = (double)i;
	}
}

__global__ void gVectorAddition(double* vector1, double* vector2, double* vectorSum, int threads_cnt) { 
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= N)
		return;
	vectorSum[i] = vector1[i] + vector2[i];
	
}

double testingThreadsOfDevice(int threads_cnt/*, double* vector1_d, double* vector2_d*/) {
	double *vectorSum_d, *vectorSum_h;
	vectorSum_h = (double*) calloc(N, sizeof(double));
	hipMalloc((void**)&vectorSum_d, N * sizeof(double)); 
	double *vector1_d, *vector2_d;
	hipMalloc((void**)&vector1_d, N * sizeof(double));
	hipMalloc((void**)&vector2_d, N * sizeof(double)); 
	gInitVectors <<< 1, 1 >>> (vector1_d, vector2_d);
	
	/// проверка: ///
	/*cudaMemcpy(vectorSum_h, vector1_d, N * sizeof(double), cudaMemcpyDeviceToHost); 
	for (int i = 0; i < N; i++)
		fprintf(stderr, "%g ", vectorSum_h[i]);
	printf("\n");
	*/
	
	struct timespec mt1, mt2;
	clock_gettime(CLOCK_REALTIME, &mt1);
	
	gVectorAddition <<< N / threads_cnt, threads_cnt >>> 
		(vector1_d, vector2_d, vectorSum_d, threads_cnt); //запуск фу-ии на GPU
	hipDeviceSynchronize(); //синхронизация потоков
	
	/// проверка: ///
	/*cudaMemcpy(vectorSum_h, vectorSum_d, N * sizeof(double), cudaMemcpyDeviceToHost); 
	for (int i = 0; i < N; i++)
		fprintf(stderr, "%g ", vectorSum_h[i]);
	printf("\n");
	*/
		
	clock_gettime(CLOCK_REALTIME, &mt2);
    	double seconds_double = (double)(mt2.tv_sec - mt1.tv_sec) +
        	(double)(mt2.tv_nsec - mt1.tv_nsec) / 1e9; ///время в секундах
	printf("blocks = %d, threads per block = %d seconds = %e \n",
		N / threads_cnt, threads_cnt, seconds_double);
	
	hipFree(vector1_d); 
	hipFree(vector2_d);
	hipFree(vectorSum_d); 
	free(vectorSum_h);
	return seconds_double;
}

int main() {
	//открытие файла для записи результатов:
	FILE *fout;
	if ((fout = fopen("output.csv", "w")) == NULL) {
		printf("error: can't open output.csv \n");
		return 1;
	}
	//инициализация векторов:
	
	//тестирование и запись в файл:
	fprintf(fout, "threads_per_block;time_in_seconds;\n");
	for (int i = 1; i <= 1024; i *= 2)//i < 100 ? i *= 2 : i += 50)
		fprintf(fout, "%d;%e;\n", i, testingThreadsOfDevice(i/*, vector1_d, vector2_d*/));

	 
	return 0;
}
