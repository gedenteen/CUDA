#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/iterator/zip_iterator.h>

#define N 32

struct rotate_tuple{
	__host__ __device__
	thrust::tuple<float,float,float> operator()(thrust::tuple<float&,float&,float&>& t) {
		float x = thrust::get<0>(t);
		float y = thrust::get<1>(t);
		float z = thrust::get<2>(t);
		
		float rx=0.36*x+0.48*y-0.80*z;
		float ry=-0.80f*x+0.60*y+0.00f*z;
		float rz=0.48f*x+0.64f*y+0.60f*z;
		return thrust::make_tuple(rx,ry,rz);
	}
};

int main(){
	thrust::device_vector<float> x(N), y(N), z(N);
	thrust::fill(x.begin(), x.end(), 2.0);
	thrust::fill(y.begin(), y.end(), 3.0);
	thrust::fill(z.begin(), z.end(), 5.0);
	
	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(x.begin(), y.begin(), z.begin() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(x.end(), y.end(), z.end() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(x.begin(), y.begin(), z.begin() )),
		rotate_tuple() 
	);
	
	thrust::host_vector<float> hx(N), hy(N), hz(N);
	hx=x; hy=y; hz=z;
	for(int i=0;i<N;i++)
		printf("%g\t%g\t%g\n",hx[i], hy[i], hz[i]);
	
	return 0;
}
