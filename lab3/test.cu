
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MyKernel(double *array, int arrayCount) 
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < arrayCount) 
  { 
    array[idx] = (double)idx; 
  } 
} 

void launchMyKernel(double *array, int arrayCount) 
{ 
  int blockSize;   // The launch configurator returned block size 
  int minGridSize; // The minimum grid size needed to achieve the 
                   // maximum occupancy for a full device launch 
  int gridSize;    // The actual grid size needed, based on input size 

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
                                      MyKernel, 0, 0); 
  // Round up according to array size 
  gridSize = (arrayCount + blockSize - 1) / blockSize; 

  MyKernel<<< gridSize, blockSize >>>(array, arrayCount); 

  hipDeviceSynchronize(); 

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 MyKernel, blockSize, 
                                                 0);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", 
         blockSize, occupancy);
}

int main(int argc, char *argv[]) {
	if (argc < 2) {
		printf("Error: run program with 1 args: array size\n");
		return 1;
	}

	double *array, arrayCount = atoi(argv[1]);
	hipMalloc((void**)&array, arrayCount * sizeof(double));
	launchMyKernel(array, arrayCount);

	return 0;
}
