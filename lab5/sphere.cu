#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI 3.14159265358979323846
#define COEF 48
#define VERTCOUNT COEF*COEF*2
#define RADIUS 10.0f
#define FGSIZE 20
#define FGSHIFT FGSIZE/2
#define IMIN(A,B) (A<B?A:B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID IMIN(32,(VERTCOUNT+THREADSPERBLOCK-1)/THREADSPERBLOCK)

typedef float(*ptr_f)(float, float, float);

struct Vertex {
	float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];
//текстура<тип, размерность текстуры, без нормализации> тексурная ссылка;
texture<float, 3, hipReadModeElementType> df_tex;
//указатель на область памяти, предназначенную для работы с текстурой:
hipArray * df_Array = 0;
//тестовая функция:
float func(float x, float y, float z) {
	return (0.5*sqrtf(15.0/M_PI))*(0.5*sqrtf(15.0/M_PI))*
		z*z*y*y*sqrtf(1.0f-z*z/RADIUS/RADIUS)/RADIUS/RADIUS
		/RADIUS/RADIUS;	
}
//проверочная фу-ия:
float check(Vertex *v, ptr_f f){
	float sum = 0.0f;
	for (int i = 0; i < VERTCOUNT; ++i)
		sum += f(v[i].x, v[i].y, v[i].z);
	return sum;
}
//дискретизация функций на прямоугольной сетке:
void calc_f(float *arr_f, int x_size, int y_size, int z_size, ptr_f f){
	for (int x = 0; x < x_size; ++x)
		for (int y = 0; y < y_size; ++y)
			for (int z = 0; z < z_size; ++z)
				arr_f[z_size * (x * y_size + y) + z] = f(x - FGSHIFT, y -
					FGSHIFT, z - FGSHIFT);
}
//определение узлов квадратуры на сфере в константной памяти.
//Котрольное вычисление квадратуры:
void init_vertices(){
	Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERTCOUNT);
	int i = 0;
	for (int iphi = 0; iphi < 2 * COEF; ++iphi){
		for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i) {
			float phi = iphi * M_PI / COEF;
			float psi = ipsi * M_PI / COEF;
			temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
			temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
			temp_vert[i].z = RADIUS * cosf(psi);
		}
	}
	printf("sumcheck = %f\n", check(temp_vert, &func)*M_PI*M_PI/
		COEF/COEF);
	//Копирует данные в 1-ый аргумент фу-ии (symbol), который находится на устройстве:
	hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) *
		VERTCOUNT, 0, hipMemcpyHostToDevice);
	free(temp_vert);
}
//копирование данных с хоста в текстуру:
void init_texture(float *df_h){
	const hipExtent volumeSize = make_hipExtent(FGSIZE,
		FGSIZE, FGSIZE);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
	hipMemcpy3DParms cpyParams={0};
	cpyParams.srcPtr = make_hipPitchedPtr( (void*)df_h,
		volumeSize.width*sizeof(float), volumeSize.width,
		volumeSize.height);
	cpyParams.dstArray = df_Array;
	cpyParams.extent = volumeSize;
	cpyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&cpyParams);
	//конфигурация текстуры, параметры фильтрации
	df_tex.normalized = false;
	df_tex.filterMode = hipFilterModeLinear;
	df_tex.addressMode[0] = hipAddressModeClamp;
	df_tex.addressMode[1] = hipAddressModeClamp;
	df_tex.addressMode[2] = hipAddressModeClamp;
	//привящка текстуры к CUDA массиву
	hipBindTextureToArray(df_tex, df_Array, channelDesc);
}
//освобожднение ресурсов:
void release_texture(){
	hipUnbindTexture(df_tex);
	hipFreeArray(df_Array);
}
//функция ядра для вычисление квадратуры:
//(кэширование фильтрованных значений функции в узлах)
__global__ void kernel(float *a)
{
	__shared__ float cache[THREADSPERBLOCK];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float x = vert[tid].x + FGSHIFT + 0.5f;
	float y = vert[tid].y + FGSHIFT + 0.5f;
	float z = vert[tid].z + FGSHIFT + 0.5f;
	cache[cacheIndex] = tex3D(df_tex, z, y, x);
	__syncthreads();
	//суммирование посредством редукции
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (cacheIndex < s)
			cache[cacheIndex] += cache[cacheIndex + s];
		__syncthreads();
	}
	if (cacheIndex == 0)
		a[blockIdx.x] = cache[0];
}
//программа-драйвер для тестирования текстурной памяти:
int main(void){
	float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
	float *sum = (float*)malloc(sizeof(float) * BLOCKSPERGRID);
	float *sum_dev;
	hipMalloc((void**)&sum_dev, sizeof(float) * BLOCKSPERGRID);
	init_vertices();//init_vertexes();
	calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);
	init_texture(arr);
	kernel<<<BLOCKSPERGRID,THREADSPERBLOCK>>>(sum_dev);
	hipDeviceSynchronize();
	hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID,
		hipMemcpyDeviceToHost);
	
	float s = 0.0f;
	for (int i = 0; i < BLOCKSPERGRID; ++i)
		s += sum[i];
	printf("sum = %f\n", s*M_PI*M_PI / COEF/COEF);
	
	hipFree(sum_dev);
	free(sum);
	release_texture();
	free(arr);
	return 0;
}
