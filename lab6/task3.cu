#include "hip/hip_runtime.h"
#include "task1.h"
#include "task2.h"

__global__ void mul_gpu(int* a, int* b, int* c) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid <= N / 256)
        c[tid] = abs(a[tid]) * abs(b[tid]) * a[tid] * b[tid] / (abs(a[tid]) * abs(b[tid]));
}
int task3() {
    //cuda-события для замерения времени выполнения:
    hipEvent_t start, stop;
    float elepsedTime;
    
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    
    printf("\n\tmultiplication of vectors through 2 streams:\n"); 
    for (long var_size = 1024; var_size <= FULL_DATA_SIZE / 2; var_size *= 2) {
        int* host_a, * host_b, * host_c;
        int* dev_a0, * dev_b0, * dev_c0;// первый gpu input buffer for stream0, который будет заполнен рандомными числами
        int* dev_a1, * dev_b1, * dev_c1;// второй gpu input buffer for stream1, который будет заполнен рандомными числами
        //выделение памяти на gpu
        hipMalloc((void**)&dev_a0, var_size * sizeof(int));
        hipMalloc((void**)&dev_b0, var_size * sizeof(int));
        hipMalloc((void**)&dev_c0, var_size * sizeof(int));
        //выделение памяти на gpu
        hipMalloc((void**)&dev_a1, var_size * sizeof(int));
        hipMalloc((void**)&dev_b1, var_size * sizeof(int));
        hipMalloc((void**)&dev_c1, var_size * sizeof(int));
        // выделение page-locked памяти, испльзуемой для стримов
        hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
        hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
        hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

        for (int i = 0; i < FULL_DATA_SIZE; i++) {
            host_a[i] = i;//rand();
            host_b[i] = i;// rand();
        }
    
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        // теперь проитерировать всю дату, через байтные куски:
        for (int i = 0; i < FULL_DATA_SIZE; i += var_size * 2) {
            // асинхронно копировать закрытую память на устройство:
            hipMemcpyAsync(dev_a0, host_a + i, var_size * sizeof(int), hipMemcpyHostToDevice, stream0);
            hipMemcpyAsync(dev_b0, host_b + i, var_size * sizeof(int), hipMemcpyHostToDevice, stream0);
            mul_gpu <<< var_size / 256, 256, 0, stream0 >>> (dev_a0, dev_b0, dev_c0);
            // копировать дату с устройства на закрытую памятьЖ
            hipMemcpyAsync(host_c + i, dev_c0, var_size * sizeof(int), hipMemcpyDeviceToHost, stream0);

            // асинхронно копировать закрытиую память на устройство 
            hipMemcpyAsync(dev_a1, host_a + i + var_size, var_size * sizeof(int), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(dev_b1, host_b + i + var_size, var_size * sizeof(int), hipMemcpyHostToDevice, stream1);
            mul_gpu <<< var_size / 256, 256, 0, stream1 >>> (dev_a1, dev_b1, dev_c1);

            // копировать дату с устройства на закрытую память
            hipMemcpyAsync(host_c + i + var_size, dev_c1, var_size * sizeof(int), hipMemcpyDeviceToHost, stream1);
        }   

        // синхронизируем оба потока
        hipStreamSynchronize(stream0);
        hipStreamSynchronize(stream1);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elepsedTime, start, stop);
        printf("var_size = FULL_DATA_SIZE / %5ld. Time taken: %f ms\n", FULL_DATA_SIZE / var_size, elepsedTime);
        
        //проверка:
        /*
        for (int i = 0; i < N / 256; i++) {
            printf("%d+%d=%d\n", host_a[i], host_b[i], host_c[i]);
        }
        */
        
        hipHostFree(host_a);
        hipHostFree(host_b);
        hipHostFree(host_c);
        hipFree(dev_a0);
        hipFree(dev_b0);
        hipFree(dev_c0);
        hipFree(dev_a1);
        hipFree(dev_b1);
        hipFree(dev_c1);
    }
    
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    return 0;
}
