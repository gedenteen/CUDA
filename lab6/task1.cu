#include "hip/hip_runtime.h"
#include "task1.h"

__global__ void gInitArray(float* arr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= M)
        return;
    arr[i] = (float)i;
}
int task1() {
    //long arr_size = 1 << 20;
    float* host_usual_arr, * host_paged_arr, * dev_arr;

    //выделение обычной памяти на хосте:
    host_usual_arr = (float*)malloc(M * sizeof(float));
    //выделение закрепленной (paged-locked) памяти на хосте:
    CUDA_CHECK_RETURN(hipHostAlloc((void**)&host_paged_arr,
        M * sizeof(float), hipHostMallocDefault));
    //выделение памяти на девайсе:
    CUDA_CHECK_RETURN(hipMalloc((void**)&dev_arr, M * sizeof(float)));

    gInitArray << < M / 128, 128 >> > (dev_arr);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    float elepsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("time of copying arrays:\n");

    hipEventRecord(start, 0);
    CUDA_CHECK_RETURN(hipMemcpy(host_paged_arr, dev_arr, M * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("device array -> host paged array = %f ms\n", elepsedTime);

    hipEventRecord(start, 0);
    CUDA_CHECK_RETURN(hipMemcpy(host_usual_arr, dev_arr, M * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("device array -> host usual array = %f ms\n", elepsedTime);

    hipEventRecord(start, 0);
    CUDA_CHECK_RETURN(hipMemcpy(dev_arr, host_paged_arr, M * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("host paged array -> device array = %f ms\n", elepsedTime);

    hipEventRecord(start, 0);
    CUDA_CHECK_RETURN(hipMemcpy(dev_arr, host_usual_arr, M * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("host usual array -> device array = %f ms\n", elepsedTime);

    printf("\n");
    free(host_usual_arr);
    hipHostFree(host_paged_arr);
    hipFree(dev_arr);
    return 0;
}
