#include "hip/hip_runtime.h"
﻿#include <stdio.h>
//#include <malloc.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

//#include "hip/hip_runtime.h"
//#include ""


#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int* a, int* b, int* c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}


__global__ void add_gpu(int* a, int *b, int* c) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid <= N/256)
        c[tid] = a[tid] + b[tid];
}

int main()
{
    hipDeviceProp_t prop;
    int whichDevice;

    // проверяем поддерживает ли устройство overlapping computation with memory copy
    // если пооддерживает  overlap, что всё хорошо 
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("device will not handle\n");
        return 0;
    }

    hipEvent_t start, stop;
    float elepsedTime;

    // создаём ивенты
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // создаём потоки 
    hipStream_t stream;
    hipStreamCreate(&stream);
    int* host_a, * host_b, * host_c;
    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    //выделение закрепленной (paged-locked) памяти:
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        kernel << < N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("Time taken: %3.1f ms\n", elepsedTime);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipStreamDestroy(stream);

    // создание ивентов
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // создание потоков
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    int* dev_a0, * dev_b0, * dev_c0;// первый gpu input buffer for stram0, который будет заполнен рандомными числами
    int* dev_a1, * dev_b1, * dev_c1;// второй gpu input buffer for stram1, который будет заполнен рандомными числами
    //выделение памяти на gpu
    hipMalloc((void**)&dev_a0, N * sizeof(int));
    hipMalloc((void**)&dev_b0, N * sizeof(int));
    hipMalloc((void**)&dev_c0, N * sizeof(int));
    //выделение памяти на gpu
    hipMalloc((void**)&dev_a1, N * sizeof(int));
    hipMalloc((void**)&dev_b1, N * sizeof(int));
    hipMalloc((void**)&dev_c1, N * sizeof(int));
    // выделение page-locked памяти, испльзуемой для стримов
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = i;//rand();
        host_b[i] = i;// rand();
    }
    // теперь проитерировать всю дату, через байтные куски
    for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
        // асинхронно копировать закрытиую память на устройство 
        hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        add_gpu << <N / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
        // копировать дату с устройства на закрытую память
        hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

        // асинхронно копировать закрытиую память на устройство 
        hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        add_gpu << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);

        // копировать дату с устройства на закрытую память
        hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }   

    // синхронизируем оба потока
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elepsedTime, start, stop);
    printf("Time taken: %3.1f ms\n", elepsedTime);

  	/*for (int i = 0; i < N / 256; i++) {
        printf("%d+%d=%d\n", host_a[i], host_b[i], host_c[i]);
    }
    */
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipStreamDestroy(stream0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipStreamDestroy(stream1);
    return 0;
}

