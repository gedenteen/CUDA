#include "header.h"

void saxpy_cublas(long int arr_size, float alpha, int iterations, 
                  hipEvent_t start, hipEvent_t stop, int check_arrays, float *time_arr)
{ 
	/// создание массивов:
	long int size_in_bytes = arr_size * sizeof(float);
	float *X_dev;
	hipMalloc((void **) &X_dev, size_in_bytes);
	float *Y_dev;
	hipMalloc((void **) &Y_dev, size_in_bytes);
	float *X_hos;
	hipHostMalloc((void **) &X_hos, size_in_bytes);
	float *Y_hos;
	hipHostMalloc((void **) &Y_hos, size_in_bytes);
	
	/// инициализация библиотеки CUBLAS:
	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&cublas_handle));
	
	/// заполнение массивов:
	for (int i=0; i < arr_size; i++){
		X_hos[i] = (float)i;
	}
	memset(Y_hos, 0, size_in_bytes);
	
	const int num_rows = arr_size; //arr_size
	const int num_cols = 1; //1
	const size_t elem_size = sizeof(float);
	
	//Копирование матрицы с числом строк arr_size и одним столбцом с
	//хоста на устройство
	hipblasSetMatrix(num_rows, num_cols, elem_size, X_hos,
		num_rows, X_dev, num_rows); //leading dimension
	hipMemset(Y_dev, 0, size_in_bytes);
	
	/// запуск SAXPY на разных размерах массивов
	const int stride = 1; //шаг (каждый stride элемент берется из массива)
	float _time; //затраченное время на SAXPY
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		hipEventRecord(start, 0);
		for (int j = 0; j < 9; j++) //saxpy вызывается несколько раз для большей точности по времени
			hipblasSaxpy(cublas_handle, tmp_size, &alpha, X_dev, stride, Y_dev, stride);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 9;
		time_arr[i * TA_COLS + 2] = _time;
		
		if (check_arrays) 
			printf("size of arrays = %ld, cuBLAS time = %f ms\n", tmp_size, _time);
	}
	
	/// проверка:
	if (check_arrays > 0) {
		hipMemcpy(X_hos, X_dev, size_in_bytes, hipMemcpyDeviceToHost);
		hipMemcpy(Y_hos, Y_dev, size_in_bytes, hipMemcpyDeviceToHost);
		for (int i = 0; i < check_arrays; i++) {
			printf("i = %d;\t X[i] = %g;\t Y[i] = %g\n", i, X_hos[i], Y_hos[i]);
		}
	}
	if (check_arrays)
		printf("\n");
	
	/// освобождение ресурсов:
	hipblasDestroy(cublas_handle);
	hipFree(X_dev);
	hipFree(Y_dev);
	hipHostFree(X_hos);
	hipHostFree(Y_hos);
}

void copying_cublas(long int arr_size, int iterations, int check_arrays,
                    hipEvent_t start, hipEvent_t stop, float *time_arr)  
{
	/// выделение памяти:
	float *host_usual_arr, *host_paged_arr, *dev1_arr, *dev2_arr;
    //выделение обычной памяти на хосте:
	long int size_in_bytes = arr_size * sizeof(float);
    host_usual_arr = (float*)malloc(size_in_bytes);
    //выделение закрепленной (paged-locked) памяти на хосте:
    hipHostAlloc((void**)&host_paged_arr, size_in_bytes, hipHostMallocDefault);
    //выделение памяти на девайсе:
    hipMalloc((void**)&dev1_arr, size_in_bytes);
    hipMalloc((void**)&dev2_arr, size_in_bytes);
	
	/// инициализация библиотеки CUBLAS:
	hipblasHandle_t cublas_handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&cublas_handle));
	
	/// заполнение массивов:
	for (int i=0; i < arr_size; i++) { //заполнить массив последовательностью
		host_usual_arr[i] = (float)i;
	}
	const int num_rows = arr_size; //arr_size
	const int num_cols = 1; //1
	const size_t elem_size = sizeof(float);
	hipblasSetMatrix(num_rows, num_cols, elem_size, host_usual_arr,
		num_rows, dev1_arr, num_rows); //leading dimension
	memset(host_usual_arr, 0, size_in_bytes); //убрать последовательность из массива, занулить
	hipMemset(dev2_arr, 0, size_in_bytes);
    	
    /// копирование массива с разными размерностями 
	const int stride = 1; //шаг (каждый stride элемент берется из массива)
	float _time; //затраченное время
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		hipEventRecord(start, 0);
    	for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			hipblasScopy(cublas_handle, tmp_size, dev1_arr, stride, dev1_arr, stride);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 5] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to device, cuBLAS time = %f ms\n", _time);
		}
		
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			hipblasGetMatrix(tmp_size, num_cols, elem_size, dev1_arr, tmp_size, host_usual_arr, tmp_size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 8] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to host usual, cuBLAS time = %f ms\n", _time);
		}
		
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			hipblasGetMatrix(tmp_size, num_cols, elem_size, dev1_arr, tmp_size, host_paged_arr, tmp_size);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 9] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to host paged, cuBLAS time = %f ms\n", _time);
		}
	}
	
	/// освобождение ресурсов:
	hipblasDestroy(cublas_handle);
	hipFree(dev1_arr);
	hipFree(dev2_arr);
	hipHostFree(host_usual_arr);
	hipHostFree(host_paged_arr);
}
