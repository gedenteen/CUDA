#include "hip/hip_runtime.h"
#include "header.h"

__global__ void saxpy(int arr_size, float alpha, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < arr_size)
		y[i] = alpha * x[i] + y[i];
}

void saxpy_cuda(long int arr_size, float alpha, int iterations, 
	hipEvent_t start, hipEvent_t stop, int check_arrays, float *time_arr)  
{
	/// создание массивов:
	long int size_in_bytes = arr_size * sizeof(float);
	float *X_dev;
	hipMalloc((void **) &X_dev, size_in_bytes);
	float *Y_dev;
	hipMalloc((void **) &Y_dev, size_in_bytes);
	float *X_hos;
	hipHostMalloc((void **) &X_hos, size_in_bytes);
	float *Y_hos;
	hipHostMalloc((void **) &Y_hos, size_in_bytes);
	
	/// заполнение массивов:
	for (int i=0; i < arr_size; i++){
		X_hos[i] = (float)i;
	}
	memset(Y_hos, 0, size_in_bytes);

	/// копирование на массивы устройства:
	hipMemcpy(X_dev, X_hos, size_in_bytes, hipMemcpyHostToDevice);
	hipMemcpy(Y_dev, Y_hos, size_in_bytes, hipMemcpyHostToDevice);

	/// запуск SAXPY на разных размерах массивов
	float _time; //затраченное время на SAXPY
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		hipEventRecord(start, 0);
		for (int j = 0; j < 9; j++) //saxpy вызывается несколько раз для большей точности по времени
		{
			saxpy <<< tmp_size / 256, 256 >>> (tmp_size, alpha, X_dev, Y_dev);
			hipDeviceSynchronize(); //синхронизация потоков
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);

		_time /= 9; //посчитать среднее время выполнения saxpy
		time_arr[i * TA_COLS] = _time; //записать время в общий массив
		
		if (check_arrays) 
			printf("size of arrays = %ld, CUDA time = %f ms\n", tmp_size, _time);
	}
	
	/// проверка:
	if (check_arrays > 0) {
		hipMemcpy(X_hos, X_dev, size_in_bytes, hipMemcpyDeviceToHost);
		hipMemcpy(Y_hos, Y_dev, size_in_bytes, hipMemcpyDeviceToHost);
		for (int i = 0; i < check_arrays; i++) {
			printf("i = %d;\t X[i] = %g;\t Y[i] = %g\n", i, X_hos[i], Y_hos[i]);
		}
	}
	if (check_arrays)
		printf("\n");
	
	/// освобождение ресурсов:
	hipFree(X_dev);
	hipFree(Y_dev);
	hipHostFree(X_hos);
	hipHostFree(Y_hos);
}

__global__ void gInitArray(long int arr_size, float* arr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= arr_size)
        return;
    arr[i] = (float)i;
}

void copying_cuda(long int arr_size, int iterations, int check_arrays,
	hipEvent_t start, hipEvent_t stop, float *time_arr) 
{
	/// создание массивов:
    float *host_usual_arr, *host_paged_arr, *dev1_arr, *dev2_arr;
	long int size_in_bytes = arr_size * sizeof(float); 
    host_usual_arr = (float*)malloc(size_in_bytes); //выделение обычной памяти на хосте
    hipHostAlloc((void**)&host_paged_arr, size_in_bytes, hipHostMallocDefault); //выделение закрепленной (paged-locked) памяти на хосте
    hipMalloc((void**)&dev1_arr, size_in_bytes); //выделение памяти на девайсе
    hipMalloc((void**)&dev2_arr, size_in_bytes);

	/// заполнение массива:
    gInitArray <<< arr_size / 256, 256 >>> (arr_size, dev1_arr); 
    hipDeviceSynchronize();
    
    /// запуск на разных размерах массивов:
	float _time; //затраченное время
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			hipMemcpy(dev2_arr, dev1_arr, tmp_size * sizeof(float), hipMemcpyDeviceToDevice);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 3] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to device, CUDA time = %f ms\n", _time);
		}
		
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
    		hipMemcpy(host_usual_arr, dev1_arr, tmp_size * sizeof(float), hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 6] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to host usual, CUDA time = %f ms\n", _time);
		}
		
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			hipMemcpy(host_paged_arr, dev1_arr, tmp_size * sizeof(float), hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 7] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to host paged, CUDA time = %f ms\n", _time);
		}
	}
	
	/// освобождение ресурсов:
	hipFree(dev1_arr);
	hipFree(dev2_arr);
	hipHostFree(host_usual_arr);
	hipHostFree(host_paged_arr);
}
