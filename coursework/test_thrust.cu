#include "header.h"

struct saxpy_functor
{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__
	float operator()(float x, float y) {
		return a * x + y;
	}
};

void saxpy(float a, thrust::device_vector<float>& x,
	thrust::device_vector<float>& y) 
{
	saxpy_functor func(a);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

void saxpy_thrust(long int arr_size, float alpha, int iterations, 
                  hipEvent_t start, hipEvent_t stop, int check_arrays, float *time_arr)
{
	/// создание и заполнение векторов векторов:
	thrust::host_vector<float> X_hos(arr_size);
	thrust::host_vector<float> Y_hos(arr_size);
	thrust::sequence(X_hos.begin(), X_hos.end());
	//thrust::fill(h2.begin(), h2.end(), 0.0);
	thrust::device_vector<float> X_dev = X_hos;
	thrust::device_vector<float> Y_dev = Y_hos;
	
	/// запуск SAXPY на разных размерах массивов
	float _time; //затраченное время на SAXPY
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		X_dev.resize(tmp_size);
		Y_dev.resize(tmp_size);
	
		hipEventRecord(start, 0);
		for (int j = 0; j < 9; j++) //saxpy вызывается несколько раз для большей точности по времени
			saxpy(alpha, X_dev, Y_dev);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 9;
		time_arr[i * TA_COLS + 1] = _time;
		
		if (check_arrays) 
			printf("size of arrays = %ld, Thrust time = %f ms\n", tmp_size, _time);
	}
	
	/// проверка:
	if (check_arrays > 0) {
		X_hos = X_dev;
		Y_hos = Y_dev;
		for (int i = 0; i < check_arrays; i++) {
			printf("i = %d;\t X[i] = %g;\t Y[i] = %g\n", i, X_hos[i], Y_hos[i]);
		}
	}
	if (check_arrays)
		printf("\n");
}

void copying_thrust(long int arr_size, int iterations, int check_arrays,
                    hipEvent_t start, hipEvent_t stop, float *time_arr)
{
	/// создание и заполнение векторов векторов:
	thrust::host_vector<float> X_hos(arr_size);
	thrust::sequence(X_hos.begin(), X_hos.end());
	thrust::device_vector<float> X_dev = X_hos;
	thrust::device_vector<float> Y_dev = X_hos;
	
	/// запуск на разных размерах массивов
	float _time; //затраченное время
	long int tmp_size = arr_size; //размер массива, который на каждой итерации уменьшаться вдвое
	for (int i = 0; i < iterations; tmp_size = tmp_size >> 1, i++) {
		X_hos.resize(tmp_size);
		X_dev.resize(tmp_size);
		Y_dev.resize(tmp_size);
	
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			Y_dev = X_dev;
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 4] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to device, Thrust time = %f ms\n", _time);
		}
		
		hipEventRecord(start, 0);
		for (int j = 0; j < 3; j++) //копирование вызывается несколько раз для большей точности по времени
			X_hos = X_dev;
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&_time, start, stop);
		
		_time /= 3;
		time_arr[i * TA_COLS + 7] = _time;
		
		if (check_arrays) {
			printf("size of arrays = %ld\n", tmp_size); 
			printf("copying device to host, Thrust time = %f ms\n", _time);
		}
	}
}

