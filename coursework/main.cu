#include "header.h"

int main(int argc, char *argv[]) {	
	/// информация об используемом устройстве:
	hipDeviceProp_t deviceProp; 
	hipGetDeviceProperties(&deviceProp, 0);
	printf("\ndevice: %s \n\n", deviceProp.name);

	/// Куда-события для замера времени:
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	/// параметры тестирования:
	int degree = 25; //степень двойки - размер массива
	float alpha = 2.5f; //коэффициент умножения для массива X
	int iterations = 6; //сколько раз запускать SAXPY
	int check_arrays = 0; //сколько элементов массива вывести на экран
	
	/// пользователь может изменить эти параметры, задав их при запуске:
	if (argc >= 2) {
		degree = atoi(argv[1]); //преобразование строки в long int
		if (degree <= 0) { 
			fprintf(stderr, "error, degree should be > 0\n");
			degree = 25;	
		}
	}
	if (argc >= 3) {
		alpha = atof(argv[2]); //преобразование строки в float
	}
	if (argc >= 4) {
		int tmp = atoll(argv[3]);
		if (tmp > 0)
			iterations = tmp;
		else
			fprintf(stderr, "error, iterations should be > 0\n");
	}
	if (argc >= 5) {
		check_arrays = atoll(argv[4]);
	}

	/// массив для записи результатов тестирования:
	long int arr_size = 1 << degree; //максимальный размер массивов
	float *time_arr = (float*) malloc(iterations * TA_COLS * sizeof(float));
	for (int i = 0; i < iterations; i++) {
		for (int j = 0; j < TA_COLS; j++) 
			time_arr[i * TA_COLS + j] = 0.0f;
	}
	
	/// посчитать среднее время и вывести результаты:
	printf("   SAXPY\n");
	saxpy_thrust(arr_size, alpha, iterations, start, stop, check_arrays, time_arr);
	saxpy_cublas(arr_size, alpha, iterations, start, stop, check_arrays, time_arr);
	saxpy_cuda(arr_size, alpha, iterations, start, stop, check_arrays, time_arr);
	
	copying_thrust(arr_size, iterations, check_arrays, start, stop, time_arr);
	copying_cuda(arr_size, iterations, check_arrays, start, stop, time_arr); //если Куду поместить выше Траста, то ломается...	
	copying_cublas(arr_size, iterations, check_arrays, start, stop, time_arr);
	
	/*
	for (int i = 0; i < iterations; i++) {
		for (int j = 0; j < TA_COLS; j++) 
			printf("%g ", time_arr[i * TA_COLS + j]);
		printf("\n");
	}*/
	
	FILE *fp;
	fp = fopen("graphs/time.csv", "w");
	if (fp == NULL) {
		fprintf(stderr, "error: can't open graphs/time.dat\n");
		exit(EXIT_FAILURE); 
	}
	///TODO: подписи к столбцам
	for (int i = 0; i < iterations; i++) {
		fprintf(fp, "%d;", degree - i);
		for (int j = 0; j < TA_COLS; j++) {
			fprintf(fp, "%g;", time_arr[i * TA_COLS + j]);
		}
		fprintf(fp, "\n");
	}
	
	fclose(fp);
	fp = fopen("graphs/ratio.csv", "w");
	if (fp == NULL) {
		fprintf(stderr, "error: can't open graphs/ratio.dat\n");
		exit(EXIT_FAILURE); 
	}
	for (int i = 0; i < iterations; i++) {
		fprintf(fp, "%d;", degree - i);
		//время saxpy-CUDA поделить на время saxpy-Thrust:
		fprintf(fp, "%g;", time_arr[i * TA_COLS] / time_arr[i * TA_COLS + 1]);
		//время saxpy-cuBLAS поделить на время saxpy-Thrust:
		fprintf(fp, "%g;", time_arr[i * TA_COLS + 2] / time_arr[i * TA_COLS + 1]);
		fprintf(fp, "\n");
	}
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}
