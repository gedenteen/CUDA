
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <malloc.h>
#define NX 64
#define BATCH 1
#define pi 3.141592

__global__ void gInitData(hipfftComplex *data){
	int i=threadIdx.x+blockDim.x*blockIdx.x;
	float x=i*2.0f*pi/(NX);
	data[i].x=cosf(x)-3.0f*sinf(x);
	data[i].y=0.0f;
}

int main(){
	//инициализация (эмуляция получения эксперементальных) данных:
	hipfftHandle plan;
	hipfftComplex *data;
	hipfftComplex *data_h=(hipfftComplex*)calloc(NX,sizeof(hipfftComplex));;
	hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;
	}
	
	gInitData<<<1, NX>>>(data);
	hipDeviceSynchronize();
	
	//конфигурация и выполнение cuFFT:
	if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return -1;
	}
	if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return -1;
	}
	if (hipDeviceSynchronize() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		return -1;
	}
	
	hipMemcpy(data_h, data, NX*sizeof(hipfftComplex),
		hipMemcpyDeviceToHost);
	
	for(int i=0;i<NX;i++)
		printf("%g\t%g\n", data_h[i].x, data_h[i].y);
	
	hipfftDestroy(plan);
	hipFree(data);
	free(data_h);
	return 0;
}
