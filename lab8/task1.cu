#include <stdio.h>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <hip/hip_runtime.h> 
#include "hipblas.h"

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error \"%s\" at line %d in file %s\n",\
			hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
} //макрос для обработки ошибок 

#define CUDA_CHECK_RETURN_CUBLAS(value) {\
	hipblasStatus_t stat = value;\
	if (stat != HIPBLAS_STATUS_SUCCESS) {\
		fprintf(stderr, "Error at line %d in file %s\n",\
			__LINE__, __FILE__);\
		exit(1);\
	}\
} //макрос для обработки ошибок 


struct saxpy_functor
{
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__
	float operator()(float x, float y) {
		return a*x+y;
	}
};
void saxpy(float a, thrust::device_vector<float>& x,
	thrust::device_vector<float>& y) 
{
	saxpy_functor func(a);
	thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}
float saxpy_thrust(long int arr_size, hipEvent_t start, hipEvent_t stop) 
{ //SAXPY с помощью thrust
	/// создание и заполнение векторов векторов:
	thrust::host_vector<float> h1(arr_size);
	thrust::host_vector<float> h2(arr_size);
	thrust::sequence(h1.begin(), h1.end());
	//thrust::fill(h2.begin(), h2.end(), 0.0);
	thrust::device_vector<float> d1 = h1;
	thrust::device_vector<float> d2 = h2;
	
	/*
	printf("before saxpy:\n");
	for (int i=0; i<16; i++) {
		printf("i = %d; h1[i]=%g; h2[i]=%g\n",i, h1[i], h2[i]);
	}
	*/
	
	hipEventRecord(start, 0);
		saxpy(2.5, d1, d2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
		
	/// вывод содержимого векторов после сложения:
	/*
	h2 = d2;
	h1 = d1;
	printf("after saxpy:\n");
	for (int i=0; i<16; i++) {
		printf("i = %d;\t h1[i] = %g;\t h2[i] = %g\n",i, h1[i], h2[i]);
	}
	*/
	
	return time;
}

float saxpy_cublas(long int arr_size, hipEvent_t start, hipEvent_t stop) 
{ //SAXPY с помощью cublas:
	const size_t size_in_bytes = (arr_size * sizeof(float));
	
	float *A_dev;
	hipMalloc( (void **) &A_dev, size_in_bytes );
	float *B_dev;
	hipMalloc( (void **) &B_dev, size_in_bytes );
	float *A_h;
	hipHostMalloc( (void **) &A_h, size_in_bytes );
	float *B_h;
	hipHostMalloc( (void **) &B_h, size_in_bytes );
	memset(A_h, 0, size_in_bytes);
	memset(B_h, 0, size_in_bytes);
	
	//инициализация библиотеки CUBLAS
	hipblasHandle_t cublas_handle;
	CUDA_CHECK_RETURN_CUBLAS(hipblasCreate(&cublas_handle));
	
	//заполнение массива А:
	for (int i=0; i < arr_size; i++){
		A_h[i] = (float)i;
	}
	/*
	printf("before saxpy (cublas):\n");
	for (int i = 0; i < 16; i++) {
		printf("i = %d;\t h1[i] = %g;\t h2[i] = %g\n", i+1, A_h[i], B_h[i]);
	}
	*/
	
	const int num_rows = arr_size / 4; //arr_size
	const int num_cols = 4; //1
	const size_t elem_size = sizeof(float);
	
	//Копирование матрицы с числом строк arr_size и одним столбцом с
	//хоста на устройство
	hipblasSetMatrix(num_rows, num_cols, elem_size, A_h,
		num_rows, A_dev, num_rows); //leading dimension
	
	//Очищаем массив на устройстве
	hipMemset(B_dev, 0, size_in_bytes);
	
	// выполнение SingleAlphaXPlusY (saxpy)
	const int stride = 1;
	float alpha = 2.5F;
	
	hipEventRecord(start, 0);
		hipblasSaxpy(cublas_handle, arr_size, &alpha, A_dev,
			stride, B_dev, stride);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	
	/*
	//Копирование матриц с числом строк arr_size и одним столбцом с
	//устройства на хост
	hipblasGetMatrix(num_rows, num_cols, elem_size, A_dev,
		num_rows, A_h, num_rows);
	hipblasGetMatrix(num_rows, num_cols, elem_size, B_dev,
		num_rows, B_h, num_rows);
	printf("after saxpy (cublas):\n");
	for (int i = 0; i < 16; i++) {
		printf("i = %d;\t h1[i] = %g;\t h2[i] = %g\n", i+1, A_h[i], B_h[i]);
	}
	*/
	
	// Освобождаем ресурсы на устройстве
	hipblasDestroy(cublas_handle);
	hipFree(A_dev);
	hipFree(B_dev);
	// Освобождаем ресурсы на хосте
	hipHostFree(A_h);
	hipHostFree(B_h);
	//сброс устройства, подготовка для выполнения новых программ
	//hipDeviceReset();
	return time;
}

int main(){	
	/// информация об используемом устройстве:
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("device: %s \n\n", deviceProp.name);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	///размерность массивов:
	const long int arr_size = 1 << 25;
	
	float time = saxpy_thrust(arr_size, start, stop);
	printf("Thrust time = %f ms\n", time);
	time = saxpy_cublas(arr_size, start, stop);
	printf("CuBLAS time = %f ms\n", time);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}
