
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <malloc.h>

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error \"%s\" at line %d in file %s\n",\
			hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
} //макрос для обработки ошибок 

void Output(float* a, int N){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++)
			fprintf(stdout,"%g\t",a[j+i*N]);
			fprintf(stdout,"\n");
		}
	fprintf(stdout,"\n\n\n");
}

__global__ void gInitializeMatrixByRows(float* matrix_d){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	matrix_d[i+j*N] = (float)(i+j*N);
}

__global__ void gInitializeMatrixByColumns(float* matrix_d){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	matrix_d[j+i*N] = (float)(j+i*N);
}

__global__ void gTranspose0(float* storage_d, float* storage_d_t){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	storage_d_t[j + i * N] = storage_d[i + j * N];
}

__global__ void gTranspose11(float* storage_d, float* storage_d_t) {
	extern __shared__ float buffer[];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	
	buffer[threadIdx.y + threadIdx.x * blockDim.y] = storage_d[i + j * N];
	__syncthreads();
	
	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	storage_d_t[i + j * N] = buffer[threadIdx.x + threadIdx.y * blockDim.x];
}

#define SH_DIM 32
__global__ void gTranspose12(float* storage_d, float* storage_d_t) {
	__shared__ float buffer_s[SH_DIM][SH_DIM];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	
	buffer_s[threadIdx.y][threadIdx.x] = storage_d[i + j * N];
	__syncthreads();
	
	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	storage_d_t[i + j * N] = buffer_s[threadIdx.x][threadIdx.y];
}

__global__ void gTranspose2(float* storage_d, float* storage_d_t) {
	__shared__ float buffer[SH_DIM][SH_DIM+1];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int N = blockDim.x * gridDim.x;
	
	buffer[threadIdx.y][threadIdx.x] = storage_d[i + j * N];
	__syncthreads();
	
	i = threadIdx.x + blockIdx.y * blockDim.x;
	j = threadIdx.y + blockIdx.x * blockDim.y;
	storage_d_t[i + j * N] = buffer[threadIdx.x][threadIdx.y];
}
	

int main(int argc, char* argv[]){
	if (argc < 3) {
		fprintf(stderr, "USAGE: matrix <dimension of matrix> <dimension_of_threads>\n");
		return -1;
	}
	int N=atoi(argv[1]);
	int dim_of_threads=atoi(argv[2]);
	if (N % dim_of_threads) {
		fprintf(stderr, "change dimensions\n");
		return -1;
	}
	int dim_of_blocks = N / dim_of_threads;
	/*
	const int max_size=1<<8;
	if (dim_of_blocks>max_size){
		fprintf(stderr, "too many blocks\n");
		return -1; 
	}*/
	
	float *storage_d, *storage_d_t, *storage_h;
	CUDA_CHECK_RETURN(hipMalloc((void**)&storage_d, N*N*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&storage_d_t, N*N*sizeof(float)));
	storage_h = (float*)calloc(N*N, sizeof(float));
	gInitializeMatrixByRows<<<dim3(dim_of_blocks,dim_of_blocks),
	dim3(dim_of_threads,dim_of_threads)>>>(storage_d);
	hipDeviceSynchronize();
	CUDA_CHECK_RETURN(hipGetLastError());
	memset(storage_h,0.0,N*N*sizeof(float));
	hipMemcpy(storage_h, storage_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
	if (argc == 4) {
		printf("изначальная матрица:\n");
		Output(storage_h, N);
	}

	gTranspose0<<<dim3(dim_of_blocks, dim_of_blocks),\
	dim3(dim_of_threads,dim_of_threads)>>>(storage_d,storage_d_t);
	hipDeviceSynchronize();
	CUDA_CHECK_RETURN(hipGetLastError());
	memset(storage_h,0.0,N*N*sizeof(float));
	hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float),
	hipMemcpyDeviceToHost);
	if (argc == 4) {
		printf("gTranspose0:\n");
		Output(storage_h, N);
	}

	gTranspose11<<<dim3(dim_of_blocks, dim_of_blocks),\
	dim3(dim_of_threads,dim_of_threads),\
	dim_of_threads*dim_of_threads*sizeof(float)>>>
	(storage_d,storage_d_t);
	hipDeviceSynchronize();
	memset(storage_h,0.0,N*N*sizeof(float));
	hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float),
	hipMemcpyDeviceToHost);
	if (argc == 4) {
		printf("gTranspose11:\n");
		Output(storage_h, N);
	}

	gTranspose12<<<dim3(dim_of_blocks, dim_of_blocks),\
	dim3(dim_of_threads,dim_of_threads)>>>
	(storage_d,storage_d_t);
	hipDeviceSynchronize();
	memset(storage_h,0.0,N*N*sizeof(float));
	hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float),
	hipMemcpyDeviceToHost);
	if (argc == 4) {
		printf("gTranspose12:\n");
		Output(storage_h, N);
	}

	gTranspose2<<<dim3(dim_of_blocks, dim_of_blocks),\
	dim3(dim_of_threads,dim_of_threads)>>>
	(storage_d,storage_d_t);
	hipDeviceSynchronize();
	memset(storage_h,0.0,N*N*sizeof(float));
	hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float),
	hipMemcpyDeviceToHost);
	if (argc == 4) {
		printf("gTranspose2:\n");
		Output(storage_h, N);
	}

	hipFree(storage_d);
	hipFree(storage_d_t);
	free(storage_h);
	return 0;
}
