#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>

/*
#define CUDA_CHECK_RETURN(value) {\
	cudaError_t _m_cudaStat = value;\
	if (_m_cudaStat != cudaSuccess) {\
		fprintf(stderr, "Error %s at line in file %s\n",\
			cudaGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	}\
}
*/

void CUDA_CHECK_RETURN(hipError_t _m_cudaStat) {
	//cudaError_t _m_cudaStat = value;
	if (_m_cudaStat != hipSuccess) {
		fprintf(stderr, "Error \"%s\" at line %d in file %s\n",
			hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);
		exit(1);
	}
}

__global__ void gTest(double* a) {
	a[threadIdx.x + blockDim.x * blockIdx.x] = 
		(double)(threadIdx.x + blockDim.x * blockIdx.x);
}

int main() {
	double *device_a, *host_a;
	int num_of_blocks = 10, threads_per_block = 1025; //error here
	int N = num_of_blocks * threads_per_block;
	
	host_a = (double*) calloc(N, sizeof(double));
	CUDA_CHECK_RETURN(hipMalloc((void**)&device_a, N * sizeof(double)));
	
	float elapsedTime;
	hipEvent_t start,stop; // встроенный тип данных – структура, для
		// фиксации контрольных точек
	hipEventCreate(&start); // инициализация
	hipEventCreate(&stop); // событий
	hipEventRecord(start,0); // привязка (регистрация) события start
	
	gTest <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (device_a);
	
	hipEventRecord(stop,0); // привязка события stop
	hipEventSynchronize(stop); // синхронизация по событию
	//CUDA_CHECK_RETURN(cudaDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
	
	hipEventElapsedTime(&elapsedTime,start,stop); // вычисление
	// затраченного времени
	fprintf(stderr,"gTest took %g\n", elapsedTime);
	hipEventDestroy(start); // освобождение
	hipEventDestroy(stop); // памяти
	
	CUDA_CHECK_RETURN(hipMemcpy(host_a, device_a, N * sizeof(double),
		hipMemcpyDeviceToHost));
	//for(int i = 0; i < N; i++)
	//	printf("%g ", host_a[i]);
		
	free(host_a);
	hipFree(device_a);
	return 0;
}
